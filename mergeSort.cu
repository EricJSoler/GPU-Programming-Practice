#include "hip/hip_runtime.h"
/*
First attempt at a parallel version of merge sort
Eric Soler 11/2015
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "gputimer.h"

#define ARRAY_SIZE 100000

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++){ 
		printf("%d ", array[i]); 
		if(i == 50) 
			printf("\n");
	}
    printf("}\n");
}

__global__ void merge(int* a,int blockSize,int arraySize){
	int s1;
	int s2; 
	int end;
	int start;
	int mid;
	start = blockSize*2*(blockIdx.x * blockDim.x + threadIdx.x);
	s1 = start;
	s2 = s1 + blockSize;
	end = s2 + blockSize;
	mid = s1 + blockSize;

	if((s2 < arraySize))
	{
		if(end > arraySize)
			end = arraySize;
	
		if(mid > arraySize)
			mid = arraySize;
	
		int* tA = new int[end - start];
		int counter = 0;

		while(counter < end - start)
		{
			if(s1 < mid && s2 < end)
			{
				if(a[s1] <= a[s2])
					tA[counter++] = a[s1++];
				else
					tA[counter++] = a[s2++];
			}
			else if(s1 < mid)
			{
				tA[counter++] = a[s1++];
			}
			else if(s2 < end)
			{
				tA[counter++] = a[s2++];
			}
			else
			{
				tA[counter++] = -66;
			}
		}
		
		for(int i = 0, j = start; i < end - start; i++, j++)
		{
			a[j] = tA[i];
		}
		delete [] tA;
	}
}

int main(int argc,char **argv)
{   
    GpuTimer timer;
	srand(time(NULL));
    // declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
	for(int i = 0; i < ARRAY_SIZE; i++)
	{
		h_array[i] =  rand()%10;
	}
	//print_array(h_array, ARRAY_SIZE);
    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 
	hipMemcpy(d_array, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);
    
	int numOfThreads;
	int blockWidth = 1000;
	int subArraySize;
	double x = log(ARRAY_SIZE) / log(2);
	timer.Start();
	int numberOfBlocks;
	for(int i = 0; i < x; i++)
	{
		subArraySize = pow(2,i);
		numOfThreads = ceil(ARRAY_SIZE/(subArraySize * 2.0));
		numberOfBlocks = ceil(numOfThreads/((float)blockWidth));
		merge<<<numberOfBlocks, ceil(numOfThreads/((float)numberOfBlocks))
				>>>(d_array, subArraySize, ARRAY_SIZE);
		hipDeviceSynchronize();
	}
    timer.Stop();
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %g ms\n", timer.Elapsed());
    hipFree(d_array);
    return 0;
}